#include "hip/hip_runtime.h"
//
// Created by binhpht on 27.3.2021.
//
#include <stdio.h>
#include "iostream"

#include "square.cuh"
__global__ void square (float * d_out, float * d_in) {
//	int idx = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	float f = d_in[idx];
	d_out[idx] = f * f;
}
void call_square (int thread_num, float * d_out, float * d_in) {
	::square<<<8,8>>>(d_out, d_in);

}